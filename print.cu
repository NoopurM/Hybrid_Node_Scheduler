
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    print_kernel<<<2, 2>>>();
    hipDeviceSynchronize();
}
