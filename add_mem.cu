//#include "../common/book.h"

#include <hip/hip_runtime.h>
#include<iostream>
#define N 10
#define CUDA_KERNEL __global__
using namespace std;

template <typename F, typename ...Args>
void cuda_launch_kernel(F fun, Args ...args) {
    fun<<<1,1>>>(args...);
  //cuda_check_last(typeid(F).name());
}

CUDA_KERNEL void add( int *a, int *b, int *c ) {
 for (int i=0; i < N; i++) {
 c[i] = a[i] + b[i];
 }
}

int main( void ) {
	 int a[N], b[N], c[N];
	 int *dev_a, *dev_b, *dev_c;

	 // allocate the memory on the GPU
	 hipMalloc( (void**)&dev_a, N * sizeof(int) ) ;
	 hipMalloc( (void**)&dev_b, N * sizeof(int) ) ;
	 hipMalloc( (void**)&dev_c, N * sizeof(int) );

	 // fill the arrays 'a' and 'b' on the CPU
	 for (int i=0; i<N; i++) {
	 a[i] = i;
	 b[i] = i * i;
	 c[i] = 0;
	 }

	// copy the arrays 'a' and 'b' to the GPU
	 hipMemcpy( dev_a, a, N * sizeof(int),
	 hipMemcpyHostToDevice );
	 hipMemcpy( dev_b, b, N * sizeof(int),
	 hipMemcpyHostToDevice ) ;
	 cuda_launch_kernel(add, dev_a, dev_b, dev_c);

	 // copy the array 'c' back from the GPU to the CPU
	 hipMemcpy( c, dev_c, N * sizeof(int),
	 hipMemcpyDeviceToHost );

	 // display the results
	 for (int i=0; i<N; i++) {
	 printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	 }

	 // free the memory allocated on the GPU
	 hipFree( dev_a );
	 hipFree( dev_b );
	 hipFree( dev_c );
	 return 0;
}
