#include <iostream>
#include <vector>
#include <queue>
#include <functional>
#include <map>
#include <chrono>
#include "mm_rec.h"
#include "scheduler.h"
using namespace std;

pthread_mutex_t sync_cnt_lock;
extern vector<pthread_t> cpu_workers;
extern vector<pthread_t> gpu_workers;

#define N 32
int m=8;

int x[N][N];
int y[N][N];
int z[N][N];

bool completed=false;

void populate_matrix(int a[N][N]) {
    for(int i=0;i<N;i++) {
        for(int j=0; j<N;j++) {
            a[i][j] = 3;
        }
    }
}

void print_matrix(int a[N][N]) {
    cout<<"Matrix :"<<endl;
    for(int i=0;i<N;i++) {
        for(int j=0; j<N;j++) {
            cout<<a[i][j]<<" ";
        }
        cout<<"\n";
    }
}

void cpu_serial_mm(int r_z, int c_z, int x[N][N], int r_x, int c_x, int y[N][N],
int r_y, int c_y, int z1[N][N], int m) {
    for(int i=r_x, u=r_z; i<r_x+m; i++,u++) {
        for(int k=c_x; k<c_x+m; k++) {
            for(int j=c_y,v=c_z ; j<c_y+m; j++,v++) {
                z[u][v] = z[u][v] + x[i][k] * y[k][j];           
            }
        }
    }
}

CUDA_KERNEL void __gpu_serial_mm__(int *r_z, int *c_z, int *x, int *r_x, int *c_x, int *y, int *r_y, int *c_y, int *dev_z, int *m) {

    for(int i=*r_x, u=*r_z; i<(*r_x+*m); i++,u++) {
        for(int k=*c_x; k<(*c_x+*m); k++) {
            for(int j=*c_y,v=*c_z ; j<(*c_y+*m); j++,v++) {	
                 dev_z[u * N + v] = dev_z[u * N + v] + x[i * N + k] * y[k * N + j]; 
            }
        }
    }
}

void input(int a[N][N], int *h_a) {
	for(int i=0; i<N; i++) {
		for(int j=0; j<N; j++) {
			h_a[i * N + j] = a[i][j];			
		}
	}
}

void output(int *h_a) {
        for(int i=0; i<N; i++) {
                for(int j=0; j<N; j++) {
                        z[i][j] = h_a[i * N + j];
                }
        }
}

void gpu_serial_mm(int r_z, int c_z, int x[N][N], int r_x, int c_x, int y[N][N], int r_y, int c_y, int z[N][N], int m) {
	int *dev_r_z, *dev_c_z, *dev_r_x, *dev_c_x, *dev_r_y, *dev_c_y, *dev_m;
	int *dev_x, *dev_y, *dev_z;
	int *h_x, *h_y, *h_z;

	hipHostMalloc((void **) &h_x, N*N*sizeof(int));
	hipHostMalloc((void **) &h_y, N*N*sizeof(int));
	hipHostMalloc((void **) &h_z, N*N*sizeof(int));
	input(x, h_x);
	input(y, h_y);
	input(z, h_z);

	hipMalloc( (void**)&dev_r_z, sizeof(int) );
	hipMalloc( (void**)&dev_c_z, sizeof(int) );
	hipMalloc( (void**)&dev_r_x, sizeof(int) );
	hipMalloc( (void**)&dev_c_x, sizeof(int) );
	hipMalloc( (void**)&dev_r_y, sizeof(int) );
	hipMalloc( (void**)&dev_c_y, sizeof(int) );
	hipMalloc( (void **)&dev_x, N*N*sizeof(int));
	hipMalloc( (void **)&dev_y, N*N*sizeof(int));
	hipMalloc( (void **)&dev_z, N*N*sizeof(int));
	hipMalloc( (void**)&dev_m, sizeof(int) );

	hipMemcpy( dev_r_z, &r_z,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( dev_c_z, &c_z,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( dev_r_x, &r_x,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( dev_c_x, &c_x,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( dev_r_y, &r_y,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( dev_c_y, &c_y,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( dev_m, &m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( dev_x, h_x, N*N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( dev_y, h_y, N*N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( dev_z, h_z, N*N*sizeof(int),hipMemcpyHostToDevice);	

	launch_kernel(__gpu_serial_mm__, dev_r_z, dev_c_z, dev_x, dev_r_x, dev_c_x, dev_y, dev_r_y, dev_c_y, dev_z, dev_m);	

	hipMemcpy( h_z, dev_z,N*N*sizeof(int),hipMemcpyDeviceToHost);
	output(h_z);
	hipFree(dev_r_z); hipFree(dev_c_z); hipFree(dev_r_x); hipFree(dev_c_x); hipFree(dev_r_y); hipFree(dev_c_y);
	hipFree(dev_x); hipFree(dev_y); hipFree(dev_z); hipFree(dev_m);
} 

void parallel_rec_mm(int r_z, int c_z, int x[N][N], int r_x, int c_x, int y[N][N], int r_y, int c_y, int n, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
    //cout<<"parallel_rec called :"<<n<<endl;
    pthread_t tid = pthread_self();
    if (n == m) {
        //z[r_z][c_z] = z[r_z][c_z] + x[r_x][c_x]*y[r_y][c_y];
        //gpu_serial_mm(r_z, c_z, x, r_x, c_x, y, r_y, c_y, z, m);
        run_task(1, cpu_serial_mm, gpu_serial_mm, r_z, c_z, x, r_x, c_x, y, r_y, c_y, z, m);
	    pthread_mutex_lock(&sync_cnt_lock);
        (*parent_sync_cnt)--;
	    pthread_mutex_unlock(&sync_cnt_lock);
    } else {
	    pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 0) {
            pthread_mutex_unlock(&sync_cnt_lock);
            int *new_child_sync_cnt1 = new int(4);
            int *child_rp1 = new int(0);
            submit_task(tid, parallel_rec_mm, r_z, c_z, x, r_x, c_x, y, r_y, c_y, n/2, child_sync_cnt, new_child_sync_cnt1, child_rp1);
            int *new_child_sync_cnt2 = new int(4);
            int *child_rp2 = new int(0);
            submit_task(tid, parallel_rec_mm, r_z, c_z+n/2, x, r_x, c_x, y, r_y, c_y+n/2, n/2, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            
            int *new_child_sync_cnt3 = new int(4);
            int *child_rp3 = new int(0);
            submit_task(tid, parallel_rec_mm, r_z+n/2, c_z, x, r_x+n/2, c_x, y, r_y, c_y, n/2, child_sync_cnt, new_child_sync_cnt3, child_rp3);
            
            int *new_child_sync_cnt4 = new int(4);
            int *child_rp4 = new int(0);
            submit_task(tid, parallel_rec_mm, r_z+n/2, c_z+n/2, x, r_x+n/2, c_x, y, r_y, c_y+n/2, n/2, child_sync_cnt, new_child_sync_cnt4, child_rp4);
            
	        pthread_mutex_lock(&sync_cnt_lock);
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_rec_mm, r_z, c_z, x, r_x, c_x, y, r_y, c_y, n, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
            pthread_mutex_unlock(&sync_cnt_lock);
        }
	    
        pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 1) {
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_rec_mm, r_z, c_z, x, r_x, c_x, y, r_y, c_y, n, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
	        pthread_mutex_unlock(&sync_cnt_lock);
        }
        
        pthread_mutex_lock(&sync_cnt_lock);
        if (*rp != 2) {
            *child_sync_cnt = 4;
	        pthread_mutex_unlock(&sync_cnt_lock);
            int *new_child_sync_cnt5 = new int(4);
            int *child_rp5 = new int(0);
            submit_task(tid, parallel_rec_mm, r_z, c_z, x, r_x, c_x+n/2, y, r_y+n/2, c_y, n/2, child_sync_cnt, new_child_sync_cnt5, child_rp5);
            
            int *new_child_sync_cnt6 = new int(4);
            int *child_rp6 = new int(0);
            submit_task(tid, parallel_rec_mm, r_z, c_z+n/2, x, r_x, c_x+n/2, y, r_y+n/2, c_y+n/2, n/2, child_sync_cnt, new_child_sync_cnt6, child_rp6);
            
            int *new_child_sync_cnt7 = new int(4);
            int *child_rp7 = new int(0);
            submit_task(tid, parallel_rec_mm, r_z+n/2, c_z, x, r_x+n/2, c_x+n/2, y, r_y+n/2, c_y, n/2, child_sync_cnt, new_child_sync_cnt7, child_rp7);
            
            int *new_child_sync_cnt8 = new int(4);
            int *child_rp8 = new int(0);
            submit_task(tid, parallel_rec_mm, r_z+n/2, c_z+n/2, x, r_x+n/2, c_x+n/2, y, r_y+n/2, c_y+n/2, n/2, child_sync_cnt, new_child_sync_cnt8, child_rp8);
        } else {
	        pthread_mutex_unlock(&sync_cnt_lock);
        }
	    
        pthread_mutex_lock(&sync_cnt_lock);
        if (*(child_sync_cnt) > 0) {
            *rp = 2;
            submit_task(tid, parallel_rec_mm, r_z, c_z, x, r_x, c_x, y, r_y, c_y, n, parent_sync_cnt, child_sync_cnt, rp);
	        pthread_mutex_unlock(&sync_cnt_lock);
            return;
        } else {
            (*parent_sync_cnt)--;
	        //cout<<"second half completed :"<<n<<endl;
            pthread_mutex_unlock(&sync_cnt_lock);
            if (n == N) {
                completed = true;
            }
        }
    }
}
int main(int argc, char *argv[]) {
    populate_matrix(x);
    populate_matrix(y);
    populate_matrix(x);
    print_matrix(x);
    print_matrix(y);
    
    create_threadpool(4);
	int *parent_sync_cnt = new int(1);
    int *rp = new int(0);
    int *child_sync_cnt = new int(4);
 
    submit_task(cpu_workers[0], parallel_rec_mm, 0, 0, x, 0, 0, y, 0, 0, N, parent_sync_cnt, child_sync_cnt, rp);
    
    wait_until_done();
    print_matrix(z);
    return 0;
}

