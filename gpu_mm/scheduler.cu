
#include <hip/hip_runtime.h>
#include <iostream>
#include <map>
#include <queue>
#include <functional>
//#include "scheduler.h"
using namespace std;

map<pthread_t, deque<function<void()>>> cpu_que_map;
map<pthread_t, pthread_mutex_t> cpu_lock_map;
map<pthread_t, deque<function<void()>>> gpu_que_map;
map<pthread_t, pthread_mutex_t> gpu_lock_map;
map<pthread_t, hipStream_t> stream_map;
extern bool completed;

pthread_t get_random_cpu() {
    static unsigned long x=123456789, y=362436069, z=521288629;

    unsigned long t;
    x ^= x << 16;
    x ^= x >> 5;
    x ^= x << 1;

    t = x;
    x = y;
    y = z;
    z = t ^ x ^ y;
    auto itr = cpu_que_map.begin();
    return next(itr, z%cpu_que_map.size())->first;
}

pthread_t get_random_gpu() {
    static unsigned long x=123456789, y=362436069, z=521288629;

    unsigned long t;
    x ^= x << 16;
    x ^= x >> 5;
    x ^= x << 1;

    t = x;
    x = y;
    y = z;
    z = t ^ x ^ y;
    auto itr = gpu_que_map.begin();
    return next(itr, z%gpu_que_map.size())->first;
}

void* __do_work_cpu(void *data) {
    pthread_t tid, tid_stolen;
    while(!completed) {
        tid = pthread_self();
        pthread_mutex_lock(&(cpu_lock_map[tid]));
        if (completed && cpu_que_map[tid].size() == 0) {
            pthread_mutex_unlock(&cpu_lock_map[tid]);
            break;
        }
        if (!cpu_que_map[tid].empty()) {
            cpu_que_map[tid].front()();
            cpu_que_map[tid].pop_front();
            pthread_mutex_unlock(&cpu_lock_map[tid]);
            continue;
        } else {
            pthread_mutex_unlock(&cpu_lock_map[tid]);
        }
        
        //Steal from other queue
        tid_stolen = get_random_cpu(); 
        pthread_mutex_lock(&cpu_lock_map[tid_stolen]); 
        if (completed && cpu_que_map[tid_stolen].size() == 0) {
            pthread_mutex_unlock(&cpu_lock_map[tid_stolen]);
            break;
        }
        if (!cpu_que_map[tid_stolen].empty()) {
            cpu_que_map[tid_stolen].back()();
            cpu_que_map[tid_stolen].pop_back();
            pthread_mutex_unlock(&cpu_lock_map[tid_stolen]);
            continue;
        } else {
            pthread_mutex_unlock(&cpu_lock_map[tid_stolen]);
        }
    }
    cout<<"CPU:"<<tid<<" exited"<<endl;
    pthread_exit(NULL);
}

void* __do_work_gpu(void *data) {
    pthread_t tid, tid_stolen;
    while(!completed) {
        tid = pthread_self();
        pthread_mutex_lock(&(gpu_lock_map[tid]));
        if (completed && gpu_que_map[tid].size() == 0) {
            pthread_mutex_unlock(&gpu_lock_map[tid]);
            break;
        }
        if (!gpu_que_map[tid].empty()) {
            gpu_que_map[tid].front()();
            gpu_que_map[tid].pop_front();
            pthread_mutex_unlock(&gpu_lock_map[tid]);
            continue;
        } else {
            pthread_mutex_unlock(&gpu_lock_map[tid]);
        }
        
        //Steal from other queue
        tid_stolen = get_random_gpu(); 
        pthread_mutex_lock(&gpu_lock_map[tid_stolen]); 
        if (completed && gpu_que_map[tid_stolen].size() == 0) {
            pthread_mutex_unlock(&gpu_lock_map[tid_stolen]);
            break;
        }
        if (!gpu_que_map[tid_stolen].empty()) {
            gpu_que_map[tid_stolen].back()();
            gpu_que_map[tid_stolen].pop_back();
            pthread_mutex_unlock(&gpu_lock_map[tid_stolen]);
            continue;
        } else {
            pthread_mutex_unlock(&gpu_lock_map[tid_stolen]);
        }
    }
    cout<<"GPU:"<<tid<<" exited"<<endl;
    pthread_exit(NULL);
}

