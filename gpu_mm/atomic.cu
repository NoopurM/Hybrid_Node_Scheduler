/*
 * Noopur Maheshwari : 111464061
 * Rahul Rane : 111465246
 */

#include <hip/hip_runtime.h>
#include <pthread.h>
#include <iostream>
using namespace std;
extern pthread_mutex_t lock;

int get_shared_var_value(int *ptr) {
    int ret;
    //cout<<"About to lock 1"<<endl;
    pthread_mutex_lock(&lock);
    //cout<<"lock 1"<<endl;
    ret = *ptr;
    //cout<<"About to unlock 1"<<endl;
    pthread_mutex_unlock(&lock);
    //cout<<"unlocked 1"<<endl;
    return ret;
}

void set_shared_var_value(int *ptr, int val) {
    //cout<<"About to lock 2"<<endl;
    pthread_mutex_lock(&lock);
    //cout<<"lock 2"<<endl;
    (*ptr) = val;
    //cout<<"About to unlock 2"<<endl;
    pthread_mutex_unlock(&lock); 
    //cout<<"unlocked 2"<<endl;
}

void dec_shared_var_value(int *ptr) {
    //cout<<"About to lock 3"<<endl;
    pthread_mutex_lock(&lock);
    //cout<<"lock 3"<<endl;
    (*ptr)--;
    //cout<<"About to unlock 3"<<endl;
    pthread_mutex_unlock(&lock); 
    //cout<<"unlocked 3"<<endl;
}
