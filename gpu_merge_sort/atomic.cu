/*
 * Noopur Maheshwari : 111464061
 * Rahul Rane : 111465246
 */

#include <hip/hip_runtime.h>
#include <pthread.h>
#include <iostream>
using namespace std;
extern pthread_mutex_t lock;

int get_shared_var_value(int *ptr) {
    int ret;
    pthread_mutex_lock(&lock);
    ret = *ptr;
    pthread_mutex_unlock(&lock);
    return ret;
}

void set_shared_var_value(int *ptr, int val) {
    pthread_mutex_lock(&lock);
    (*ptr) = val;
    pthread_mutex_unlock(&lock); 
}

void dec_shared_var_value(int *ptr) {
    pthread_mutex_lock(&lock);
    (*ptr)--;
    pthread_mutex_unlock(&lock); 
}
