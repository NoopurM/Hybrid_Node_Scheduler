
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <map>
using namespace std;

extern void *do_work(void *data);
extern map<pthread_t, pthread_mutex_t> lock_map;
vector<pthread_t> workers;

bool create_threadpool(int nworkers) {
    workers.resize(nworkers);
    bool ret = true;
    for (int i=0; i<nworkers; i++) {
        if(pthread_create(&workers[i], NULL, do_work, (void *)(i+1))) {
            cout<<"Failed to create thread for id :"<<i<<endl;
            ret = false;
        } else {
            pthread_mutex_init(&lock_map[workers[i]], NULL);
        }
    }
    return ret;
}

void wait_until_done() {
    void *status;
    for(int i=0; i<workers.size(); i++) {
        if(pthread_join(workers[i], &status)) {
            cout<<"Failed to join thread for id :"<<i<<endl;
        }
    }
    return;
}
