
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <map>
using namespace std;

extern void *__do_work_cpu(void *data);
extern void *__do_work_gpu(void *data);
extern map<pthread_t, pthread_mutex_t> cpu_lock_map;
extern map<pthread_t, pthread_mutex_t> gpu_lock_map;
extern map<pthread_t, hipStream_t> stream_map;
vector<pthread_t> cpu_workers;
vector<pthread_t> gpu_workers;

bool __create_cpu_threadpool(int nworkers) {
    cpu_workers.resize(nworkers);
    bool ret = true;
    for (int i=0; i<nworkers; i++) {
        if(pthread_create(&cpu_workers[i], NULL, __do_work_cpu, (void *)(i+1))) {
            cout<<"Failed to create thread for id :"<<i<<endl;
            ret = false;
        } else {
            pthread_mutex_init(&cpu_lock_map[cpu_workers[i]], NULL);
        }
    }
    return ret;
}

bool __create_gpu_threadpool(int nworkers) {
    gpu_workers.resize(nworkers);
    bool ret = true;
    for (int i=0; i<nworkers; i++) {
        if(pthread_create(&gpu_workers[i], NULL, __do_work_gpu, (void *)(i+1))) {
            //cout<<"Failed to create thread for id :"<<i<<endl;
            ret = false;
        } else {
            hipStreamCreate(&stream_map[gpu_workers[i]]);
            pthread_mutex_init(&gpu_lock_map[gpu_workers[i]], NULL);
        }
    }
    return ret;
}

bool create_threadpool(int nworkers) {
    bool ret;
    ret = __create_cpu_threadpool(nworkers);
    if (!ret) {
        cout<<"Failed to spawn cpu threads"<<endl;
    }

    ret = __create_gpu_threadpool(nworkers);
    if (!ret) {
        //cout<<"Failed to spawn gpu threads"<<endl;
    }

    return ret;
}

void wait_until_done() {
    void *status;
    for(int i=0; i<cpu_workers.size(); i++) {
        if(pthread_join(cpu_workers[i], &status)) {
            cout<<"Failed to join thread for id :"<<i<<endl;
        }
    }
    cout<<"CPU threads finished"<<endl; 
    for(int i=0; i<gpu_workers.size()-1; i++) {
        if(pthread_join(gpu_workers[i], &status)) {
            cout<<"Failed to join thread for id :"<<i<<endl;
        }
    }
    hipDeviceReset();
    cout<<"GPU threads finished"<<endl;
    return;
}
