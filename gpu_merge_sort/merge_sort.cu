#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cilk/cilk.h>
#include <cilk/cilk_api.h>
#include "merge_sort.h"
#include <vector>
#include <map>
#include <queue>
#include <functional>
using namespace std;

extern map<pthread_t, deque<function<void()>>> que_map;
pthread_mutex_t sync_cnt_lock;
extern vector<pthread_t> workers;

bool completed=false;
//int arr[10] = {23,12,11,33,2,1,4,22,12,10};
int arr[15] = {15,14,13,12,11,10,9,8,7,6,5,4,3,2,1};

template < typename PTHREADID, typename CALLABLE, typename... ARGS >
void submit_task(PTHREADID tid, CALLABLE fn, ARGS&&... args ) { 
    //cout<<"Submitting task to "<<tid<<" queue"<<endl; 
    que_map[tid].push_back( bind( fn, args... ) ) ; 
}

//void merge(int *arr, int p, int q, int r) {
void merge(int p, int q, int r) {
	int left[q-p+1], right[r-q];
	int i,j,k;
	for(i=0;i<q-p+1;i++) {
		left[i] = arr[i+p];
	}
	for(i=0;i<r-q;i++) {
		right[i] = arr[i+q+1];
	}
	for(k=p,i=0,j=0;i<(q-p+1) && j<(r-q);) {
		if (left[i] <= right[j]) {
			arr[k] = left[i];
			i++;
			k++;
		} else {
			arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(q-p+1)) {
		arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(r-q)) {
		arr[k] = right[j];
		k++;
		j++;
	}
}
//void parallel_merge_sort(int *arr, int p, int r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
void parallel_merge_sort(int p, int r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
	pthread_t tid = pthread_self();
    cout<<"parallel merge sort called :"<<p<<" "<<r<<endl;
    if (p < r) {
		int q = floor((p+r)/2);
		pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 0) {
            pthread_mutex_unlock(&sync_cnt_lock);
            //int *new_child_sync_cnt1 = new int(2);
            int *new_child_sync_cnt1;
	    hipMallocManaged(&new_child_sync_cnt1, sizeof(int));
	    *new_child_sync_cnt1 = 2;

            //int *child_rp1 = new int(0);
	    int *child_rp1;
	    hipMallocManaged(&child_rp1, sizeof(int));
	    *child_rp1 = 0;
            //submit_task(tid, parallel_merge_sort, arr, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
            submit_task(tid, parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
            //cilk_spawn parallel_merge_sort(arr, p, q);
		    
            //int *new_child_sync_cnt2 = new int(2);
	    int *new_child_sync_cnt2;
            hipMallocManaged(&new_child_sync_cnt2, sizeof(int));
            *new_child_sync_cnt2 = 2;

            //int *child_rp2 = new int(0);
	    int *child_rp2;
            hipMallocManaged(&child_rp2, sizeof(int));
            *child_rp2 = 0; 

            //submit_task(tid, parallel_merge_sort, arr, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            submit_task(tid, parallel_merge_sort, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            //parallel_merge_sort(arr, q+1, r);
		    //cilk_sync;
            
            pthread_mutex_lock(&sync_cnt_lock);
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                //submit_task(tid, parallel_merge_sort, arr, p, r, parent_sync_cnt, child_sync_cnt, rp);
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
            pthread_mutex_unlock(&sync_cnt_lock);
        }
	    
        pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 1) {
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                //submit_task(tid, parallel_merge_sort, arr, p, r, parent_sync_cnt, child_sync_cnt, rp);
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
	        pthread_mutex_unlock(&sync_cnt_lock);
        }

		//merge(arr, p, q, r);
		merge(p, q, r);
        pthread_mutex_lock(&sync_cnt_lock);
		(*parent_sync_cnt)--;
	    cout<<"second half completed :"<<r-p<<endl;
        pthread_mutex_unlock(&sync_cnt_lock);
        if ((r-p) == 14) {
            completed = true;
        }
	} else {
        pthread_mutex_lock(&sync_cnt_lock);
        (*parent_sync_cnt)--;
	    pthread_mutex_unlock(&sync_cnt_lock);
    }
}
int main() {
	//int arr[10] = {23,12,11,33,2,1,4,22,12,10};
	//parallel_merge_sort(arr, 0, 9);
	create_threadpool(4);
	//int *parent_sync_cnt = new int(2);
	int *parent_sync_cnt;
	hipMallocManaged(&parent_sync_cnt, sizeof(int));
	*parent_sync_cnt = 2;	

    	//int *rp = new int(0);
	int *rp;
	hipMallocManaged(&rp, sizeof(int));
	*rp = 0;		

    	//int *child_sync_cnt = new int(2);
	int *child_sync_cnt;
	hipMallocManaged(&child_sync_cnt, sizeof(int));
	*child_sync_cnt = 2;
 
    //submit_task(workers[0], parallel_merge_sort, arr, 0, 9, parent_sync_cnt, child_sync_cnt, rp);
    submit_task(workers[0], parallel_merge_sort, 0, 14, parent_sync_cnt, child_sync_cnt, rp);
    
    wait_until_done();
    
    for (int i=0;i<15;i++) {
		cout<<arr[i]<<" ";
	}
}
