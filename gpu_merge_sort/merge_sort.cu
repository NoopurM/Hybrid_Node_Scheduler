#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "merge_sort.h"
#include <vector>
#include <map>
#include <queue>
#include <functional>
#include "scheduler.h"
using namespace std;

pthread_mutex_t sync_cnt_lock;
extern vector<pthread_t> workers;

bool completed=false;
#define N 15
int arr[N];

__global__ void __gpu_merge__(int *d_arr, int *p, int *q, int *r) {
	printf("########## %d %d %d", *p, *q, *r);
    int left_n = *q-*p+1;
    int right_n = *r-*q;
	int *left = new int[left_n];
    int *right = new int[right_n];
    int i,j,k;
	for(i=0;i<*q-*p+1;i++) {
		left[i] = d_arr[i+*p];
	}
	for(i=0;i<*r-*q;i++) {
		right[i] = d_arr[i+*q+1];
	}
	for(k=*p,i=0,j=0;i<(*q-*p+1) && j<(*r-*q);) {
		if (left[i] <= right[j]) {
			d_arr[k] = left[i];
			i++;
			k++;
		} else {
			d_arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(*q-*p+1)) {
		d_arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(*r-*q)) {
		d_arr[k] = right[j];
		k++;
		j++;
	}
    delete[] left;
    delete[] right;
}

void cpu_merge(int p, int q, int r) {
    int left[q-p+1], right[r-q];
    int i,j,k;
	for(i=0;i<q-p+1;i++) {
		left[i] = arr[i+p];
	}
	for(i=0;i<r-q;i++) {
		right[i] = arr[i+q+1];
	}
	for(k=p,i=0,j=0;i<(q-p+1) && j<(r-q);) {
		if (left[i] <= right[j]) {
			arr[k] = left[i];
			i++;
			k++;
		} else {
			arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(q-p+1)) {
		arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(r-q)) {
		arr[k] = right[j];
		k++;
		j++;
	}
}

void gpu_merge(int p, int q, int r) {
    int *d_p, *d_r, *d_q, *d_arr;
    cout<<"Executing merge on GPU"<<endl;
    hipMalloc((void **)&d_p, sizeof(int));
    hipMalloc((void **)&d_q, sizeof(int));
    hipMalloc((void **)&d_r, sizeof(int));
    hipMalloc((void **)&d_arr, N*sizeof(int));
    hipMemcpy( d_p, &p, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_q, &q, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_r, &r, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    launch_kernel(__gpu_merge__, d_arr, d_p, d_q, d_r);
    hipMemcpy(arr, d_arr, N * sizeof(int),hipMemcpyDeviceToHost);
    hipFree(d_p); hipFree(d_q);hipFree(d_r); hipFree(d_arr); 
}

void parallel_merge_sort(int p, int r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
	pthread_t tid = pthread_self();
    //cout<<"parallel merge sort called :"<<p<<" "<<r<<endl;
    if (p < r) {
        int q;
		q = floor((p+r)/2);
        
		pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 0) {
            pthread_mutex_unlock(&sync_cnt_lock);
	        int *new_child_sync_cnt1 = new int(2);
            int *child_rp1 = new int(0);
            submit_task(tid, parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
	        
            int *new_child_sync_cnt2 = new int(2);
            int *child_rp2 = new int(0);
            submit_task(tid, parallel_merge_sort, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            
            pthread_mutex_lock(&sync_cnt_lock);
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
            pthread_mutex_unlock(&sync_cnt_lock);
        }
	    
        pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 1) {
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
	        pthread_mutex_unlock(&sync_cnt_lock);
        }

        //gpu_merge(p, q, r);
        //cpu_merge(p, q, r);
        run_task(0, cpu_merge, gpu_merge, p, q, r);
        pthread_mutex_lock(&sync_cnt_lock);
		(*parent_sync_cnt)--;
	    //cout<<"second half completed :"<<r-p<<endl;
        pthread_mutex_unlock(&sync_cnt_lock);
        if ((r-p) == N-1) {
            completed = true;
        }
	} else {
        pthread_mutex_lock(&sync_cnt_lock);
        (*parent_sync_cnt)--;
	    pthread_mutex_unlock(&sync_cnt_lock);
    }
}
int main() {
	create_threadpool(4);
    for(int i=N-1;i>=0;i--) {
	    arr[i] = rand()%15;
    }

    for(int i=0;i<N;i++) {
        cout<<"arr :"<<arr[i]<<endl;
    }
	int *parent_sync_cnt = new int(2);
    int *rp = new int(0);
    int *child_sync_cnt = new int(2);
 
    submit_task(workers[0], parallel_merge_sort, 0, N-1, parent_sync_cnt, child_sync_cnt, rp);
    
    wait_until_done();
    
    for (int i=0;i<N;i++) {
		cout<<arr[i]<<" ";
	}
}
