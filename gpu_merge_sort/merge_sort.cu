#include "hip/hip_runtime.h"
#include "merge_sort.h"

pthread_mutex_t lock;
extern vector<pthread_t> cpu_workers;
extern vector<pthread_t> gpu_workers;

bool completed=false;
#define N 15
int arr[N];

__global__ void __gpu_merge__(int *d_arr, int *p, int *q, int *r) {
	printf("########## %d %d %d", *p, *q, *r);
    int left_n = *q-*p+1;
    int right_n = *r-*q;
	int *left = new int[left_n];
    int *right = new int[right_n];
    int i,j,k;
	for(i=0;i<*q-*p+1;i++) {
		left[i] = d_arr[i+*p];
	}
	for(i=0;i<*r-*q;i++) {
		right[i] = d_arr[i+*q+1];
	}
	for(k=*p,i=0,j=0;i<(*q-*p+1) && j<(*r-*q);) {
		if (left[i] <= right[j]) {
			d_arr[k] = left[i];
			i++;
			k++;
		} else {
			d_arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(*q-*p+1)) {
		d_arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(*r-*q)) {
		d_arr[k] = right[j];
		k++;
		j++;
	}
    delete[] left;
    delete[] right;
}

void cpu_merge(int p, int q, int r, int *sync_cnt) {
    int left[q-p+1], right[r-q];
    int i,j,k;
	for(i=0;i<q-p+1;i++) {
		left[i] = arr[i+p];
	}
	for(i=0;i<r-q;i++) {
		right[i] = arr[i+q+1];
	}
	for(k=p,i=0,j=0;i<(q-p+1) && j<(r-q);) {
		if (left[i] <= right[j]) {
			arr[k] = left[i];
			i++;
			k++;
		} else {
			arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(q-p+1)) {
		arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(r-q)) {
		arr[k] = right[j];
		k++;
		j++;
	}
    dec_shared_var_value(sync_cnt);
}

void gpu_merge(int p, int q, int r, int *sync_cnt) {
    int *d_p, *d_r, *d_q, *d_arr;
    cout<<"Executing merge on GPU"<<endl;
    hipMalloc((void **)&d_p, sizeof(int));
    hipMalloc((void **)&d_q, sizeof(int));
    hipMalloc((void **)&d_r, sizeof(int));
    hipMalloc((void **)&d_arr, N*sizeof(int));
    hipMemcpy( d_p, &p, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_q, &q, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_r, &r, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    launch_kernel(__gpu_merge__, d_arr, d_p, d_q, d_r);
    hipMemcpy(arr, d_arr, N * sizeof(int),hipMemcpyDeviceToHost);
    hipFree(d_p); hipFree(d_q);hipFree(d_r); hipFree(d_arr);
    dec_shared_var_value(sync_cnt);
}

void parallel_merge_sort(int p, int r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
	pthread_t tid = pthread_self();
    int local_sync_cnt, local_rp;
    //cout<<"parallel merge sort called :"<<p<<" "<<r<<endl;
    if (p < r) {
        int q;
		q = floor((p+r)/2);
        
        local_rp = get_shared_var_value(rp);
        if (local_rp == 0) {
	        int *new_child_sync_cnt1 = new int(2);
            int *child_rp1 = new int(0);
            submit_task(tid, parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
	        
            int *new_child_sync_cnt2 = new int(2);
            int *child_rp2 = new int(0);
            submit_task(tid, parallel_merge_sort, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            
            set_shared_var_value(rp, 1);
            submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
            return;
        }
	    
        local_rp = get_shared_var_value(rp);
        if (local_rp == 1) {
            local_sync_cnt = get_shared_var_value(child_sync_cnt);
            if (local_sync_cnt > 0) {
                set_shared_var_value(rp, 1);
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
                return;
            }
            cout<<"Setting rp to 2"<<endl;
            set_shared_var_value(rp, 2);
        }

        //gpu_merge(p, q, r);
        //cpu_merge(p, q, r);
        local_rp = get_shared_var_value(rp);
        if (local_rp == 2) {
            set_shared_var_value(child_sync_cnt, 1);
            run_task(0, cpu_merge, gpu_merge, p, q, r, child_sync_cnt);
            set_shared_var_value(rp, 3);
            submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
            return;
        }
        
        local_rp = get_shared_var_value(rp);
        if (local_rp == 3) {
            local_sync_cnt = get_shared_var_value(child_sync_cnt);
            if (local_sync_cnt > 0) {
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
                return;
            } 
	    }

        dec_shared_var_value(parent_sync_cnt);
	    cout<<"second half completed :"<<r-p<<endl;
        if ((r-p) == N-1) {
            completed = true;
        }
	} else {
        dec_shared_var_value(parent_sync_cnt);
        if ((r-p) == N-1) {
            completed = true;
        }
    }
}
int main() {
    bool ret;	
    ret = create_threadpool(4);
    if (!ret) {
        cout<<"Failed to create threadpool"<<endl;
        //return -1;
    }
    for(int i=N-1;i>=0;i--) {
	    arr[i] = rand()%15;
    }

    for(int i=0;i<N;i++) {
        cout<<"arr :"<<arr[i]<<endl;
    }
	int *parent_sync_cnt = new int(2);
    int *rp = new int(0);
    int *child_sync_cnt = new int(2);
 
    submit_task(cpu_workers[0], parallel_merge_sort, 0, N-1, parent_sync_cnt, child_sync_cnt, rp);
    
    wait_until_done();
    
    for (int i=0;i<N;i++) {
		cout<<arr[i]<<" ";
	}
}
