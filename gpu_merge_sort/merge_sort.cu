#include "hip/hip_runtime.h"
/*
 * Noopur Maheshwari : 111464061
 * Rahul Rane : 111465246
 */
#include <chrono>
#include "merge_sort.h"

pthread_mutex_t lock;
extern vector<pthread_t> cpu_workers;
extern vector<pthread_t> gpu_workers;

bool completed=false;
/*
 * For tasks to be running on CPU - 0
 * for tasks to be running on GPU - 1
 * for tasks to be running on CPU/GPU - 2
 */
#define RUN_FLAG 0
#define N 16

int arr[N];

__global__ void __gpu_merge__(int *d_arr, int *p, int *q, int *r) {
    int left_n = *q-*p+1;
    int right_n = *r-*q;
	int *left = new int[left_n];
    int *right = new int[right_n];
    int i,j,k;
	for(i=0;i<*q-*p+1;i++) {
		left[i] = d_arr[i+*p];
	}
	for(i=0;i<*r-*q;i++) {
		right[i] = d_arr[i+*q+1];
	}
	for(k=*p,i=0,j=0;i<(*q-*p+1) && j<(*r-*q);) {
		if (left[i] <= right[j]) {
			d_arr[k] = left[i];
			i++;
			k++;
		} else {
			d_arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(*q-*p+1)) {
		d_arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(*r-*q)) {
		d_arr[k] = right[j];
		k++;
		j++;
	}
    delete[] left;
    delete[] right;
}

void cpu_merge(int p, int q, int r, int *sync_cnt) {
    cout<<"Executing merge on CPU"<<endl;
    int left[q-p+1], right[r-q];
    int i,j,k;
	for(i=0;i<q-p+1;i++) {
		left[i] = arr[i+p];
	}
	for(i=0;i<r-q;i++) {
		right[i] = arr[i+q+1];
	}
	for(k=p,i=0,j=0;i<(q-p+1) && j<(r-q);) {
		if (left[i] <= right[j]) {
			arr[k] = left[i];
			i++;
			k++;
		} else {
			arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(q-p+1)) {
		arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(r-q)) {
		arr[k] = right[j];
		k++;
		j++;
	}
    dec_shared_var_value(sync_cnt);
}

void gpu_merge(int p, int q, int r, int *sync_cnt) {
    int *d_p, *d_r, *d_q, *d_arr;
    cout<<"Executing merge on GPU"<<endl;
    hipMalloc((void **)&d_p, sizeof(int));
    hipMalloc((void **)&d_q, sizeof(int));
    hipMalloc((void **)&d_r, sizeof(int));
    hipMalloc((void **)&d_arr, N*sizeof(int));
    hipMemcpy( d_p, &p, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_q, &q, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_r, &r, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    launch_kernel(__gpu_merge__, d_arr, d_p, d_q, d_r);
    hipMemcpy(arr, d_arr, N * sizeof(int),hipMemcpyDeviceToHost);
    hipFree(d_p); hipFree(d_q);hipFree(d_r); hipFree(d_arr);
    dec_shared_var_value(sync_cnt);
}

void parallel_merge_sort(int p, int r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
	pthread_t tid = pthread_self();
    int local_sync_cnt, local_rp;
    if (p < r) {
        int q;
		q = floor((p+r)/2);
        
        local_rp = get_shared_var_value(rp);
        if (local_rp == 0) {
	        int *new_child_sync_cnt1 = new int(2);
            int *child_rp1 = new int(0);
            submit_task(tid, parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
	        
            int *new_child_sync_cnt2 = new int(2);
            int *child_rp2 = new int(0);
            submit_task(tid, parallel_merge_sort, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            
            set_shared_var_value(rp, 1);
            submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
            return;
        }
	    
        local_rp = get_shared_var_value(rp);
        if (local_rp == 1) {
            local_sync_cnt = get_shared_var_value(child_sync_cnt);
            if (local_sync_cnt > 0) {
                set_shared_var_value(rp, 1);
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
                return;
            }
            set_shared_var_value(rp, 2);
        }

        local_rp = get_shared_var_value(rp);
        if (local_rp == 2) {
            set_shared_var_value(child_sync_cnt, 1);
            run_task(RUN_FLAG, cpu_merge, gpu_merge, p, q, r, child_sync_cnt);
            set_shared_var_value(rp, 3);
            submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
            return;
        }
        
        local_rp = get_shared_var_value(rp);
        if (local_rp == 3) {
            local_sync_cnt = get_shared_var_value(child_sync_cnt);
            if (local_sync_cnt > 0) {
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
                return;
            } 
	    }

        dec_shared_var_value(parent_sync_cnt);
        if ((r-p) == N-1) {
            completed = true;
        }
	} else {
        dec_shared_var_value(parent_sync_cnt);
        if ((r-p) == N-1) {
            completed = true;
        }
    }
}
int main() {
    create_threadpool(4);
    init_bayes();
    for(int i=N-1;i>=0;i--) {
	    arr[i] = rand()%N;
    }

    cout<<"Input array :"<<endl;
    for(int i=0;i<N;i++) {
        cout<<arr[i]<<" ";
    }
    cout<<endl;
	int *parent_sync_cnt = new int(2);
    int *rp = new int(0);
    int *child_sync_cnt = new int(2);
    
    chrono::time_point<std::chrono::system_clock> start, end;
    start = chrono::system_clock::now();
 
    submit_task(cpu_workers[0], parallel_merge_sort, 0, N-1, parent_sync_cnt, child_sync_cnt, rp);
    
    wait_until_done();
    
    end = chrono::system_clock::now();
    chrono::duration<double> elapsed_seconds = end - start;
    cout<<"Sorted array :"<<endl; 
    for (int i=0;i<N;i++) {
		cout<<arr[i]<<" ";
	}
    cout<<endl;
    cout<<"Run flag :"<<RUN_FLAG<<endl;
    cout<<"N :"<<N<<endl;
    cout<<"Job time = "<<elapsed_seconds.count()<<"seconds"<<endl;
}
