#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "merge_sort.h"
#include <vector>
#include <map>
#include <queue>
#include <functional>
#include "scheduler.h"
using namespace std;

pthread_mutex_t sync_cnt_lock;
extern vector<pthread_t> workers;

bool completed=false;
#define N 15
int arr[N];
int *d_arr;

//void merge(int *arr, int p, int q, int r) {
__global__ void merge(int *arr, int *p, int *q, int *r) {
	int left_n = *q-*p+1;
    int right_n = *r-*q;
    //int left[left_n], right[right_n];
	int *left = new int[left_n];
    int *right = new int[right_n];
    int i,j,k;
	for(i=0;i<*q-*p+1;i++) {
		left[i] = arr[i+*p];
	}
	for(i=0;i<*r-*q;i++) {
		right[i] = arr[i+*q+1];
	}
	for(k=*p,i=0,j=0;i<(*q-*p+1) && j<(*r-*q);) {
		if (left[i] <= right[j]) {
			arr[k] = left[i];
			i++;
			k++;
		} else {
			arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(*q-*p+1)) {
		arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(*r-*q)) {
		arr[k] = right[j];
		k++;
		j++;
	}
    delete[] left;
    delete[] right;
}

void parallel_merge_sort(int p, int r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
	pthread_t tid = pthread_self();
    cout<<"parallel merge sort called :"<<p<<" "<<r<<endl;
    if (p < r) {
        int q;
		q = floor((p+r)/2);
        
		pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 0) {
            pthread_mutex_unlock(&sync_cnt_lock);
	        int *new_child_sync_cnt1 = new int(2);
            int *child_rp1 = new int(0);
            submit_task(tid, parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
	        
            int *new_child_sync_cnt2 = new int(2);
            int *child_rp2 = new int(0);
            submit_task(tid, parallel_merge_sort, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            
            pthread_mutex_lock(&sync_cnt_lock);
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
            pthread_mutex_unlock(&sync_cnt_lock);
        }
	    
        pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 1) {
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
	        pthread_mutex_unlock(&sync_cnt_lock);
        }

	    int *d_p, *d_r, *d_q;
        hipMalloc((void **)&d_p, sizeof(int));
        hipMalloc((void **)&d_q, sizeof(int));
        hipMalloc((void **)&d_r, sizeof(int));
	    hipMemcpy( d_p, &p, sizeof(int), hipMemcpyHostToDevice);
	    hipMemcpy( d_q, &q, sizeof(int), hipMemcpyHostToDevice);
	    hipMemcpy( d_r, &r, sizeof(int), hipMemcpyHostToDevice);
	    hipMemcpy( d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
	    launch_kernel(merge, d_arr, d_p, d_q, d_r);
	    //merge<<<1,1>>>(d_arr, d_p, d_q, d_r);
        hipMemcpy( arr, d_arr, N * sizeof(int),hipMemcpyDeviceToHost);
        hipFree(d_p); hipFree(d_q);hipFree(d_r); hipFree(d_arr); 
        pthread_mutex_lock(&sync_cnt_lock);
		(*parent_sync_cnt)--;
	    cout<<"second half completed :"<<r-p<<endl;
        pthread_mutex_unlock(&sync_cnt_lock);
        if ((r-p) == 14) {
            completed = true;
        }
	} else {
        pthread_mutex_lock(&sync_cnt_lock);
        (*parent_sync_cnt)--;
	    pthread_mutex_unlock(&sync_cnt_lock);
    }
}
int main() {
	create_threadpool(4);
    for(int i=14;i>=0;i--) {
	    arr[i] = i+1;
    }

    for(int i=0;i<N;i++) {
        cout<<"arr :"<<arr[i]<<endl;
    }
	int *parent_sync_cnt = new int(2);
    int *rp = new int(0);
    int *child_sync_cnt = new int(2);
 
    submit_task(workers[0], parallel_merge_sort, 0, N-1, parent_sync_cnt, child_sync_cnt, rp);
    
    wait_until_done();
    
    for (int i=0;i<15;i++) {
		cout<<arr[i]<<" ";
	}
}
