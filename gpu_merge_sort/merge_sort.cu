#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "merge_sort.h"
#include <vector>
#include <map>
#include <queue>
#include <functional>
#include "scheduler.h"
using namespace std;

pthread_mutex_t sync_cnt_lock;
extern vector<pthread_t> workers;

bool completed=false;
#define N 15
//int arr[15];
int *arr;

//void merge(int *arr, int p, int q, int r) {
void merge(int *arr, int *p, int *q, int *r) {
	int left_n = *q-*p+1;
    int right_n = *r-*q;
    //int left[left_n], right[right_n];
	int *left = new int[left_n];
    int *right = new int[right_n];
    int i,j,k;
	for(i=0;i<*q-*p+1;i++) {
		left[i] = arr[i+*p];
	}
	for(i=0;i<*r-*q;i++) {
		right[i] = arr[i+*q+1];
	}
	for(k=*p,i=0,j=0;i<(*q-*p+1) && j<(*r-*q);) {
		if (left[i] <= right[j]) {
			arr[k] = left[i];
			i++;
			k++;
		} else {
			arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(*q-*p+1)) {
		arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(*r-*q)) {
		arr[k] = right[j];
		k++;
		j++;
	}
    delete[] left;
    delete[] right;
}
//void parallel_merge_sort(int *arr, int p, int r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
void parallel_merge_sort(int *p, int *r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
	pthread_t tid = pthread_self();
    cout<<"parallel merge sort called :"<<*p<<" "<<*r<<endl;
    if (*p < *r) {
        int *q;
        hipMallocManaged((void **)&q, sizeof(int));
		*q = floor((*p+*r)/2);
        
		//int q = floor((p+r)/2);
		pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 0) {
            pthread_mutex_unlock(&sync_cnt_lock);
            int *new_child_sync_cnt1, *child_rp1;
	        hipMallocManaged((void **)&new_child_sync_cnt1, sizeof(int));
	        hipMallocManaged((void **)&child_rp1, sizeof(int));
	        
            *new_child_sync_cnt1 = 2;
	        *child_rp1 = 0;
            //submit_task(tid, parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
            submit_task(tid, parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
            //launch_kernel(parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
 
	        int *new_child_sync_cnt2, *child_rp2;
            hipMallocManaged((void **)&new_child_sync_cnt2, sizeof(int));
            hipMallocManaged((void **)&child_rp2, sizeof(int));
            *new_child_sync_cnt2 = 2;
            *child_rp2 = 0; 
            //submit_task(tid, parallel_merge_sort, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            int *q1;
            hipMallocManaged((void **)&q1, sizeof(int));
            *q1 = *q+1;
            submit_task(tid, parallel_merge_sort, q1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            //launch_kernel(parallel_merge_sort, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            
            pthread_mutex_lock(&sync_cnt_lock);
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
                //launch_kernel(parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
            pthread_mutex_unlock(&sync_cnt_lock);
        }
	    
        pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 1) {
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
                //launch_kernel(parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
	        pthread_mutex_unlock(&sync_cnt_lock);
        }

		launch_kernel(merge, arr, p, q, r);
        pthread_mutex_lock(&sync_cnt_lock);
		(*parent_sync_cnt)--;
	    cout<<"second half completed :"<<*r-*p<<endl;
        pthread_mutex_unlock(&sync_cnt_lock);
        if ((*r-*p) == 14) {
            completed = true;
        }
	} else {
        pthread_mutex_lock(&sync_cnt_lock);
        (*parent_sync_cnt)--;
	    pthread_mutex_unlock(&sync_cnt_lock);
    }
}
int main() {
	create_threadpool(4);
    hipMallocManaged((void **)&arr, 25*sizeof(int));
    //hipMalloc((void **)&d_arr, N*sizeof(int));
    for(int i=14;i>=0;i--) {
        //hipMallocManaged(&arr[i], sizeof(int));
	    arr[i] = i+1;
    }

    for(int i=0;i<15;i++) {
        cout<<"arr :"<<arr[i]<<endl;
    }
	int *parent_sync_cnt, *child_sync_cnt, *rp;
	hipMallocManaged((void **)&parent_sync_cnt, sizeof(int));
	hipMallocManaged((void **)&rp, sizeof(int));
	hipMallocManaged((void **)&child_sync_cnt, sizeof(int));
	*parent_sync_cnt = 2;	
	*rp = 0;		
	*child_sync_cnt = 2;
 
    int *p, *r;
    hipMallocManaged((void **)&p, sizeof(int));
    hipMallocManaged((void **)&r, sizeof(int));
    *p = 0;
    *r = 14;
    submit_task(workers[0], parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
    
    wait_until_done();
    
    for (int i=0;i<15;i++) {
		cout<<arr[i]<<" ";
	}
}
