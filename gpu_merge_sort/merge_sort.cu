#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "merge_sort.h"
#include <vector>
#include <map>
#include <queue>
#include <functional>
using namespace std;

extern map<pthread_t, deque<function<void()>>> que_map;
pthread_mutex_t sync_cnt_lock;
extern vector<pthread_t> workers;

bool completed=false;
//int arr[10] = {23,12,11,33,2,1,4,22,12,10};
//int arr[15] = {15,14,13,12,11,10,9,8,7,6,5,4,3,2,1};
int *arr;
/*hipMallocManaged(&arr, 15*sizeof(int));
for(int i=14;i>=0;i++) {
    //hipMallocManaged(&arr[i], sizeof(int));
	arr[i] = i+1;
}*/

template < typename CALLABLE, typename ...ARGS >
__global__ void launch_task(ARGS ...args) {
    //printf("IN KERNEL : Launching task");
    CALLABLE()(args...);
    //printf("IN KERNEL : Launched task");
}

template < typename CALLABLE, typename ...ARGS >
void launch_kernel(CALLABLE fn, ARGS ...args) {
    cout<<"About to launch task"<<endl;
    launch_task<CALLABLE, ARGS...><<<1,1>>>(args...);
    cout<<"launched kernel"<<endl;
    hipDeviceSynchronize();
}

template < typename PTHREADID, typename CALLABLE, typename... ARGS >
void submit_task(PTHREADID tid, CALLABLE fn, ARGS&&... args ) { 
    cout<<"Submitting task to "<<tid<<" queue"<<endl; 
    que_map[tid].push_back( bind( fn, args... ) ) ; 
}

//void merge(int *arr, int p, int q, int r) {
__global__ void merge(int *arr, int *p, int *q, int *r) {
	int left_n = *q-*p+1;
    int right_n = *r-*q;
    //int left[left_n], right[right_n];
	int *left = new int[left_n];
    int *right = new int[right_n];
    int i,j,k;
	for(i=0;i<*q-*p+1;i++) {
		left[i] = arr[i+*p];
	}
	for(i=0;i<*r-*q;i++) {
		right[i] = arr[i+*q+1];
	}
	for(k=*p,i=0,j=0;i<(*q-*p+1) && j<(*r-*q);) {
		if (left[i] <= right[j]) {
			arr[k] = left[i];
			i++;
			k++;
		} else {
			arr[k] = right[j];
			j++;
			k++;
		}
	}
	while(i<(*q-*p+1)) {
		arr[k] = left[i];
		k++;
		i++;
	}
	while(j<(*r-*q)) {
		arr[k] = right[j];
		k++;
		j++;
	}
    delete[] left;
    delete[] right;
}
//void parallel_merge_sort(int *arr, int p, int r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
void parallel_merge_sort(int *p, int *r, int *parent_sync_cnt, int *child_sync_cnt, int *rp) {
	pthread_t tid = pthread_self();
    cout<<"parallel merge sort called :"<<*p<<" "<<*r<<endl;
    if (*p < *r) {
        int *q;
        hipMallocManaged((void **)&q, sizeof(int));
		*q = floor((*p+*r)/2);
        
		//int q = floor((p+r)/2);
		pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 0) {
            pthread_mutex_unlock(&sync_cnt_lock);
            int *new_child_sync_cnt1, *child_rp1;
	        hipMallocManaged((void **)&new_child_sync_cnt1, sizeof(int));
	        hipMallocManaged((void **)&child_rp1, sizeof(int));
	        
            *new_child_sync_cnt1 = 2;
	        *child_rp1 = 0;
            //submit_task(tid, parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
            submit_task(tid, parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
            //launch_kernel(parallel_merge_sort, p, q, child_sync_cnt, new_child_sync_cnt1, child_rp1);
 
	        int *new_child_sync_cnt2, *child_rp2;
            hipMallocManaged((void **)&new_child_sync_cnt2, sizeof(int));
            hipMallocManaged((void **)&child_rp2, sizeof(int));
            *new_child_sync_cnt2 = 2;
            *child_rp2 = 0; 
            //submit_task(tid, parallel_merge_sort, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            int *q1;
            hipMallocManaged((void **)&q1, sizeof(int));
            *q1 = *q+1;
            submit_task(tid, parallel_merge_sort, q1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            //launch_kernel(parallel_merge_sort, q+1, r, child_sync_cnt, new_child_sync_cnt2, child_rp2);
            
            pthread_mutex_lock(&sync_cnt_lock);
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
                //launch_kernel(parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
            pthread_mutex_unlock(&sync_cnt_lock);
        }
	    
        pthread_mutex_lock(&sync_cnt_lock);
        if (*rp == 1) {
            if (*(child_sync_cnt) > 0) {
                *rp = 1;
                submit_task(tid, parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
                //launch_kernel(parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
	            pthread_mutex_unlock(&sync_cnt_lock);
                return;
            }
	        pthread_mutex_unlock(&sync_cnt_lock);
        } else {
	        pthread_mutex_unlock(&sync_cnt_lock);
        }

		//merge(arr, p, q, r);
		//launch_kernel(merge, arr, p, q, r);
		//launch_kernel(merge, p, *q, r);
        merge<<<1,1>>>(arr, p, q, r);
        hipDeviceSynchronize();
        pthread_mutex_lock(&sync_cnt_lock);
		(*parent_sync_cnt)--;
	    cout<<"second half completed :"<<*r-*p<<endl;
        pthread_mutex_unlock(&sync_cnt_lock);
        if ((*r-*p) == 14) {
            completed = true;
        }
	} else {
        pthread_mutex_lock(&sync_cnt_lock);
        (*parent_sync_cnt)--;
	    pthread_mutex_unlock(&sync_cnt_lock);
    }
}
int main() {
	create_threadpool(4);
    hipMallocManaged((void **)&arr, 25*sizeof(int));
    for(int i=14;i>=0;i--) {
        //hipMallocManaged(&arr[i], sizeof(int));
	    arr[i] = i+1;
    }
    for(int i=0;i<15;i++) {
        cout<<"arr :"<<arr[i]<<endl;
    }
	int *parent_sync_cnt, *child_sync_cnt, *rp;
	hipMallocManaged((void **)&parent_sync_cnt, sizeof(int));
	hipMallocManaged((void **)&rp, sizeof(int));
	hipMallocManaged((void **)&child_sync_cnt, sizeof(int));
	*parent_sync_cnt = 2;	
	*rp = 0;		
	*child_sync_cnt = 2;
 
    int *p, *r;
    hipMallocManaged((void **)&p, sizeof(int));
    hipMallocManaged((void **)&r, sizeof(int));
    *p = 0;
    *r = 14;
    submit_task(workers[0], parallel_merge_sort, p, r, parent_sync_cnt, child_sync_cnt, rp);
    
    wait_until_done();
    
    for (int i=0;i<15;i++) {
		cout<<arr[i]<<" ";
	}
}
